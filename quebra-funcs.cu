#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <crypt.h>

char vetor[66] = "./1234567890qwertyuiopasdfghjklzxcvbnmQWERTYUIOPASDFGHJKLZXCVBNM"; //64
__shared__ char palavraVetor1[64][2];
__shared__ char palavraVetor2[128][3];

__global__ void func1() {
	int i;
	char palavra[3] = "\0\0";
	char *cp;

	//palavraVetor = (char*) malloc(sizeof(palavra)*64);
	for (i = 0; i < 64; ++i){
		palavra[0] = vetor[i];
		*cp = *palavraVetor1[i];
		while (*cp++ = *palavra++);
		//strcpy(palavraVetor1[i],palavra);
		//printf("%s\n", palavraVetor1[i]);
	}
}

int main(int argc, char const *argv[])
{
	
	char *result;
	int ok,i;
	char salt[15] = "Fk9Rzlrnj.fqg"; //q
	//char salt[15] = "Fk99B8qn1jmV2"; //qq
	//char salt[15] = "FksK2de5SVsUM"; //qqq
	//char salt[15] = "FkLwp/xodTQRE"; //qqqq
	//char salt[15] = "Fk/lj89JcV0Ic"; //qqqqq
	
	for ( i = 0; i < 64; ++i){
		result = crypt(palavraVetor1[i], "Fk");
		ok = strcmp (result, salt) == 0;
		if (ok) {
			printf("%s\n", palavraVetor1[i]);
			exit(0);
		}
	}
/*
	for (temp1 = 0; temp1 < 63; ++temp1)	{
		palavra[19] = vetor[temp1];
		for (temp2 = 0; temp2 < 63; ++temp2){
			palavra[18] = vetor[temp2];
			for (temp3 = 0; temp3 < 63; ++temp3){
				palavra[17] = vetor[temp3];
				for (temp4 = 0; temp4 < 63; ++temp4){
					palavra[16] = vetor[temp4];
					for (temp5 = 0; temp5 < 63; ++temp5){
						palavra[15] = vetor[temp5];
						for (temp6 = 0; temp6 < 63; ++temp6){
							palavra[14] = vetor[temp6];
							for (temp7 = 0; temp7 < 63; ++temp7){
								palavra[13] = vetor[temp7];
								for (temp8 = 0; temp8 < 63; ++temp8){
									palavra[12] = vetor[temp8];
									for (temp9 = 0; temp9 < 63; ++temp9){
										palavra[11] = vetor[temp9];
										for (temp10 = 0; temp10 < 63; ++temp10){
											palavra[10] = vetor[temp10];
											for (temp11 = 0; temp11 < 63; ++temp11){
												palavra[9] = vetor[temp11];
												for (temp12 = 0; temp12 < 63; ++temp12){
													palavra[8] = vetor[temp12];
													for (temp13 = 0; temp13 < 63; ++temp13){
														palavra[7] = vetor[temp13];
														for (temp14 = 0; temp14 < 63; ++temp14){
															palavra[6] = vetor[temp14];
															for (temp15 = 0; temp15 < 63; ++temp15){
																palavra[5] = vetor[temp15];
																for (temp16 = 0; temp16 < 63; ++temp16){
																	palavra[4] = vetor[temp16];
																	for (temp17 = 0; temp17 < 63; ++temp17){
																		palavra[3] = vetor[temp17];
																		for (temp18 = 0; temp18 < 63; ++temp18){
																			palavra[2] = vetor[temp18];
																			for (temp19 = 0; temp19 < 63; ++temp19){
																				palavra[1] = vetor[temp19];
																				for (temp20 = 0; temp20 < 63; ++temp20){
																					palavra[0] = vetor[temp20];
																					printf("%s\n", palavra);
																					result = __host__ crypt(palavra, "Fk");
																					ok = strcmp (result, salt) == 0;
																					if (ok) {
																						printf("%s\n", palavra);
																						exit(1);
																					}
																					if (flag1 == 0) {
																						palavra[1] = '\0';
																						flag1 = 1;
																					}
																				}
																				if (flag2 == 0) {
																					palavra[2] = '\0';
																					flag2 = 1;
																				}								
																			}
																			if (flag3 == 0) {
																				palavra[3] = '\0';
																				flag3 = 1;
																			}
																		}
																		if (flag4 == 0) {
																			palavra[4] = '\0';
																			flag4 = 1;
																		}
																	}
																	if (flag5 == 0) {
																		palavra[5] = '\0';
																		flag5 = 1;
																	}
																}
																if (flag6 == 0) {
																	palavra[6] = '\0';
																	flag6 = 1;
																}
															}
															if (flag7 == 0) {
																palavra[7] = '\0';
																flag7 = 1;
															}
														}
														if (flag8 == 0) {
															palavra[8] = '\0';
															flag8 = 1;
														}
													}
													if (flag9 == 0) {
														palavra[9] = '\0';
														flag9 = 1;
													}
												}
												if (flag10 == 0) {
													palavra[10] = '\0';
													flag10 = 1;
												}
											}
											if (flag11 == 0) {
												palavra[11] = '\0';
												flag11 = 1;
											}
										}
										if (flag12 == 0) {
											palavra[12] = '\0';
											flag12 = 1;
										}
									}
									if (flag13 == 0) {
										palavra[13] = '\0';
										flag13 = 1;
									}
								}
								if (flag14 == 0) {
									palavra[14] = '\0';
									flag14 = 1;
								}
							}
							if (flag15 == 0) {
								palavra[15] = '\0';
								flag15 = 1;
							}
						}
						if (flag16 == 0) {
							palavra[16] = '\0';
							flag16 = 1;
						}
					}
					if (flag17 == 0) {
						palavra[17] = '\0';
						flag17 = 1;
					}
				}
				if (flag18 == 0) {
					palavra[18] = '\0';
					flag18 = 1;
				}
			}
			if (flag19 == 0) {
				palavra[19] = '\0';
				flag19 = 1;
			}
		}
		if (flag20 == 0) {
			palavra[20] = '\0';
			flag20 = 1;
		}
	}
	*/
	return 0;
}